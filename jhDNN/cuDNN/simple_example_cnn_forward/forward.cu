#include <iostream>
#include <cstdlib>
#include <hipDNN.h>


#define cudnnErrChk(ans) { cudnnAssert((ans), __FILE__, __LINE__); }
inline void cudnnAssert(hipdnnStatus_t code, const char *file, int line, bool abort=true)
{
   if (code != HIPDNN_STATUS_SUCCESS) 
   {
      fprintf(stderr,"cuDNN assert: %s %s %d\n", hipdnnGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#define cudaErrChk(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"CUDA assert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}



int main(int argc, char** argv) {

    hipdnnHandle_t cudnn;
    cudnnErrChk (hipdnnCreate (&cudnn));

    /***
     *** 1. Describing operands : Input, output tensors and filter (kernel, weights)
     ***/
    
    // Input tensor
    hipdnnTensorDescriptor_t input_desc;
    cudnnErrChk (hipdnnCreateTensorDescriptor (&input_desc));
    cudnnErrChk (hipdnnSetTensor4dDescriptor (input_desc, 
        /*LAYOUT*/HIPDNN_TENSOR_NCHW, /*DATATYPE*/HIPDNN_DATA_FLOAT, /*N*/1, /*C*/ 3, /*H*/128, /*W*/128));

    // Output tensor
    hipdnnTensorDescriptor_t output_desc;
    cudnnErrChk (hipdnnCreateTensorDescriptor (&output_desc));
    cudnnErrChk (hipdnnSetTensor4dDescriptor (output_desc, 
        /*LAYOUT*/HIPDNN_TENSOR_NCHW, /*DATATYPE*/HIPDNN_DATA_FLOAT, /*N*/1, /*C*/ 3, /*H*/128, /*W*/128));

    // Kernel == Filter == Weights
    hipdnnFilterDescriptor_t kernel_desc;
    cudnnErrChk (hipdnnCreateFilterDescriptor (&kernel_desc));
    cudnnErrChk (hipdnnSetFilter4dDescriptor (kernel_desc, 
        /*DATATYPE*/HIPDNN_DATA_FLOAT, /*LAYOUT*/HIPDNN_TENSOR_NCHW, /*O_C*/3, /*I_C*/3, /*K_H*/3, /*K_W*/3));

    
    /***
     *** 2. Describing the convolution kernel
     ***/
    
    // Convolution layer
    hipdnnConvolutionDescriptor_t convolution_desc;
    cudnnErrChk (hipdnnCreateConvolutionDescriptor (&convolution_desc));
    cudnnErrChk (hipdnnSetConvolution2dDescriptor (convolution_desc, 
        /*PAD_H*/1, /*PAD_W*/1, /*STRIDE_VERTICAL*/1, /*STRIDE_HORIZONTAL*/1, /*DILATION_H*/1, /*DILATION_W*/1, /*MODE*/HIPDNN_CROSS_CORRELATION, /*DATATYPE*/HIPDNN_DATA_FLOAT));

    // Convolution algorithm
    hipdnnConvolutionFwdAlgo_t convolution_algo;
    cudnnErrChk (hipdnnGetConvolutionForwardAlgorithm (cudnn, input_desc, kernel_desc, convolution_desc, output_desc, HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &convolution_algo));

    /***
     *** 3. Get work space size
     ***/
    size_t bytes_workspace = 0;
    cudnnErrChk (hipdnnGetConvolutionForwardWorkspaceSize (cudnn, input_desc, kernel_desc, convolution_desc, output_desc, convolution_algo, &bytes_workspace));
    printf("Workspace size: %.3f MB\n", ((float)bytes_workspace)*1e-6);
 

    /***
     *** 4. Allocating memories
     ***/
    void *d_workspace = nullptr;
    cudaErrChk (hipMalloc (&d_workspace, bytes_workspace));
    float *h_input=nullptr, *h_output=nullptr, *h_kernel=nullptr;
    float *d_input=nullptr, *d_output=nullptr, *d_kernel=nullptr;

    h_input = (float*) malloc (1*3*128*128*sizeof(float));
    h_output = (float*) malloc (1*3*128*128*sizeof(float));
    h_kernel = (float*) malloc (3*3*3*3*sizeof(float));
    cudaErrChk (hipMalloc (&d_input, 1*3*128*128*sizeof(float)));
    cudaErrChk (hipMalloc (&d_output, 1*3*128*128*sizeof(float)));
    cudaErrChk (hipMalloc (&d_kernel, 3*3*3*3*sizeof(float)));

    /***
     *** 5. Launch the kernel
     ***/
    const float alpha=1, beta=0;
    cudnnErrChk (hipdnnConvolutionForward (cudnn
                                        , /*ALPHA*/&alpha
                                        , /*INPUT*/input_desc, d_input
                                        , /*KERNEL*/kernel_desc, d_kernel
                                        , /*LAYER*/convolution_desc, convolution_algo, d_workspace, bytes_workspace
                                        , /*BETA*/&beta
                                        , /*OUTPUT*/output_desc, d_output
                                    ))


    /***
     *** 6. Get the result
     ***/
    cudaErrChk (hipMemcpy (h_output, d_output, 1*3*128*128*sizeof(float), hipMemcpyDeviceToHost));



    /*** 
     *** 7. Finalize : Free all objects
     ***/
    free (h_input);
    free (h_output);
    free (h_kernel);

    cudaErrChk (hipFree (d_workspace));
    cudaErrChk (hipFree (d_input));
    cudaErrChk (hipFree (d_output));
    cudaErrChk (hipFree (d_kernel));

    cudnnErrChk (hipdnnDestroyTensorDescriptor (input_desc));
    cudnnErrChk (hipdnnDestroyTensorDescriptor (output_desc));
    cudnnErrChk (hipdnnDestroyFilterDescriptor (kernel_desc));
    cudnnErrChk (hipdnnDestroyConvolutionDescriptor (convolution_desc));
    cudnnErrChk (hipdnnDestroy(cudnn));
    return 0;
}
