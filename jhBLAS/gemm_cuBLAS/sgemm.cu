

#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <random>
#include <sys/time.h>
#include <hipblas.h>


static const char *_cudaGetErrorEnum(hipblasStatus_t error)
{
    switch (error)
    {
        case HIPBLAS_STATUS_SUCCESS:
            return "CUBLAS_STATUS_SUCCESS";

        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "CUBLAS_STATUS_NOT_INITIALIZED";

        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "CUBLAS_STATUS_ALLOC_FAILED";

        case HIPBLAS_STATUS_INVALID_VALUE:
            return "CUBLAS_STATUS_INVALID_VALUE";

        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "CUBLAS_STATUS_ARCH_MISMATCH";

        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "CUBLAS_STATUS_MAPPING_ERROR";

        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "CUBLAS_STATUS_EXECUTION_FAILED";

        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "CUBLAS_STATUS_INTERNAL_ERROR";
    }

    return "<unknown>";
}

#define cuBLASErrChk(ans) { cuBLASAssert((ans), __FILE__, __LINE__); }
inline void cuBLASAssert(hipblasStatus_t code, const char *file, int line, bool abort=true)
{
   if (code != HIPBLAS_STATUS_SUCCESS) 
   {
      fprintf(stderr,"CUDA assert: %s %s %d\n", _cudaGetErrorEnum(code), file, line);
      if (abort) exit(code);
   }
}
#define cudaErrChk(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"CUDA assert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

struct config {
    int AH;
    int AW;
    int BH;
    int BW;
    int CH;
    int CW;
    float alpha;
    float beta;
    bool do_test;
};


/***************************************
  * Device code "matmul"
  **************************************/


/***************************************
  * Host code "matmul"
  **************************************/
float* host_mat_mul(const float* A, const float* B, const float* C, const struct config conf) {

    printf("[Kernel] Run kernal\n");
    /*** Initialize device memory ***/
    size_t size_A = sizeof(float)*conf.AH*conf.AW;
    size_t size_B = sizeof(float)*conf.BH*conf.BW;
    size_t size_C = sizeof(float)*conf.CH*conf.CW;
    float *d_A, *d_B, *d_C;
    float *result = (float *) malloc (conf.CH*conf.CW*sizeof(float));
    cudaErrChk (hipMalloc ((void**)(&d_A), size_A));
    cudaErrChk (hipMalloc ((void**)(&d_B), size_B));
    cudaErrChk (hipMalloc ((void**)(&d_C), size_C));
    cudaErrChk (hipMemcpy (d_A, A, size_A, hipMemcpyHostToDevice));
    cudaErrChk (hipMemcpy (d_B, B, size_B, hipMemcpyHostToDevice));
    cudaErrChk (hipMemcpy (d_C, C, size_C, hipMemcpyHostToDevice));
    cudaErrChk (hipDeviceSynchronize ())

    /*** Setup execution config ***/
    hipblasHandle_t handle;
    cuBLASErrChk (hipblasCreate (&handle));
    int m=conf.CH, n=conf.CW, k=conf.AW;
    const float *alpha=&(conf.alpha), *beta=&(conf.beta);
    /*** Run CUDA kernel ***/
    hipEvent_t start, stop;
    cudaErrChk(hipEventCreate(&start));
    cudaErrChk(hipEventCreate(&stop));
    cudaErrChk(hipEventRecord(start, NULL));
    // Main body
    cuBLASErrChk (hipblasSgemm (handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, alpha, d_B, n, d_A, k, beta, d_C, n));

    // End of main body
    cudaErrChk(hipEventRecord(stop, NULL));
    cudaErrChk(hipEventSynchronize(stop));
    float msec_total = 0.0f;
    float gflo = conf.CH*conf.CW*(2.0*conf.AW+2)*1e-9;
    cudaErrChk(hipEventElapsedTime(&msec_total, start, stop));
    printf("    Elaped time: %.4f msec\n", msec_total);
    printf("    gFlops : %.4f gflops\n", gflo/(msec_total*1e-3));

    cudaErrChk (hipMemcpy(result, d_C, size_C, hipMemcpyDeviceToHost));
    cudaErrChk (hipDeviceSynchronize ())
    cudaErrChk (hipFree (d_A));
    cudaErrChk (hipFree (d_B));
    cudaErrChk (hipFree (d_C));
    cuBLASErrChk (hipblasDestroy (handle));

    return result;
}



/****************************************
  * Helper functions for host
  ****************************************/

const struct config host_get_cmd_args(int argc, char** argv) {

    int a=100, b=100, c=100;
    float alpha=1.0f, beta=0.0f;
    bool do_test = false;

    if (argc >= 2)
        do_test = (bool)atoi(argv[1]);
    if (argc >= 7) {
        a = atoi(argv[2]);
        b = atoi(argv[3]);
        c = atoi(argv[4]);
        alpha = atof(argv[5]);
        beta = atof(argv[6]);
    }

    struct config conf = {
        a,
        b,
        b,
        c,
        a,
        c,
        alpha,
        beta,
        do_test
    };
    printf("\n================================================\n");
    printf("CUDA implementaion of SGEMM\n");
    printf("    args: ./matmul [test] [a, b, c, alpha, beta]\n");
    printf("    C[a, c] = alpha * A[a, b] @ B[b, c] + beta * C[a, c]\n");
    printf("    C[%d, %d] = %f * A[%d, %d] @ B[%d, %d] + %f * C[%d, %d]\n", a,c,alpha,a,b,b,c,beta,a,c);
    printf("================================================\n\n");
    return conf;
}


void host_initialize(float *mem, const int H, const int W) {
    for (int i=0; i<H; i++) {
        for (int j=0; j<W; j++) {
            mem[i*W+j] = (float)(rand()%100);
        }
    }
}

void host_test(const float *A, const float *B, const float *C, const float * result, const struct config conf) {

    if (conf.do_test == false) {
        printf("[TEST] Test skipped..\n");
        return;
    }

    printf("[TEST] Test start..\n");

    float alpha=conf.alpha, beta=conf.beta;
    int len_k = conf.AW;
    for (int i=0; i<conf.CH; i++) {
        for (int j=0; j<conf.CW; j++) {
            float sum = 0;
            for (int k=0; k<len_k; k++) {
                sum += A[i*conf.AW+k]*B[k*conf.BW+j];
            }
            sum = alpha*sum+beta*C[i*conf.CW+j];
            if (sum != result[i*conf.CW+j]){
                printf("    [ERROR] C[%d][%d] = %.f != %f\n", i, j, result[i*conf.CW+j], sum);
                printf("    Test failed...!\n");
                return;
            }
        }
    }
    printf("    Test passed!!\n");
    return;
}


/***************************************
  * Main function
  **************************************/
int main(int argc, char** argv) {

    /*** Program configuration ***/
    const struct config conf = host_get_cmd_args(argc, argv);
    srand(0);

    /*** Initialize Data ***/
    float *A = (float *) malloc (conf.AH*conf.AW*sizeof(float));
    float *B = (float *) malloc (conf.BH*conf.BW*sizeof(float));
    float *C = (float *) calloc (conf.CH*conf.CW,sizeof(float));
    host_initialize(A, conf.AH, conf.AW);
    host_initialize(B, conf.BH, conf.BW);
    host_initialize(C, conf.CH, conf.CW);
    size_t total_size = (size_t)(conf.AH*conf.AW*sizeof(float) + conf.BH*conf.BW*sizeof(float) + 2.0*conf.CH*conf.CW*sizeof(float));
    printf("[Mem] Total size of matrices : %.3fGB\n", total_size*1e-9);

    /*** Run matmul ***/
    float* result = host_mat_mul (A, B, C, conf); 

    /*** Test result ***/
    host_test(A, B, C, result, conf);

    /*** Finalize ***/
    free (A);
    free (B);
    free (C);
    free (result);

    return 0;
}


