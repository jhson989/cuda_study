

#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>

#define DTYPE unsigned long long
#define ull unsigned long long

/*** CUDA API error checking  ***/
#define cudaErrChk(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"CUDA assert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}



/********************************
  *** Kernel program
  ********************************/

__global__ void reduction (DTYPE* d_data, ull remain, ull next) {
    ull idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx + next < remain) {
        d_data[idx] += d_data[idx+next];
    }
}







/********************************
  *** Host program
  ********************************/


DTYPE initial_data (DTYPE* data, const ull num_data) {
    DTYPE sum = 0;
    for (ull i=0; i<num_data; i++) {
        data[i] = rand()%5-2;
        sum += data[i];
    }
    return sum;
}




int main (int argc, char** argv) {

    /*** Program Configuration ***/
    const ull num_data = 5*1e+8;
    const size_t size_data = sizeof (ull) * num_data;
    printf("\n\n=======================================================================\n");
    printf("== Parallel integer reduction\n");
    printf("=======================================================================\n");
    printf("Number of int : %llu\n", num_data);
    printf("    size of mem : %.2f GB\n", size_data*1e-9);


    /*** Initialize variables ***/
    DTYPE* data = (DTYPE*) malloc (size_data);
    const DTYPE sum = initial_data (data, num_data);



    /*** Set CUDA Memory ***/
    DTYPE* d_data;
    cudaErrChk (hipMalloc ((void**)&d_data, size_data));
    cudaErrChk (hipMemcpy (d_data, data, size_data, hipMemcpyHostToDevice));

    /*** Run kernel ***/
    ull remain=num_data, next=0;
    while (remain > 1e+3) {
        if (remain%2==0)
            next = remain/2;
        else
            next = remain/2 +1;

        dim3 threads (256);
        dim3 blocks ((next+threads.x-1)/threads.x);
        reduction<<<blocks, threads>>> (d_data, remain, next);
        cudaErrChk (hipDeviceSynchronize ())
        cudaErrChk (hipGetLastError() );
        remain = next;
    }



    /*** Check result ***/
    DTYPE* result = (DTYPE*) malloc (sizeof (DTYPE)*remain);
    cudaErrChk (hipMemcpy (result, d_data, sizeof (DTYPE)*remain, hipMemcpyDeviceToHost));
    for (int i=1; i<remain; i++)
        result[0] += result[i];


    printf("Check result ...\n");
    if (sum != result[0]) {
        printf("Err GT(%llu) != Pred(%llu)\n", sum, result[0]);
    } else {
        printf("Pass GT(%llu) == Pred(%llu)\n", sum, result[0]);
    }
    printf("=======================================================================\n\n");

    /*** Finalize program ***/
    cudaErrChk (hipFree (d_data));    
    free (data);
    free (result);

    return 0;
}
