#include "hip/hip_runtime.h"

#include <cstdio>
#include <cstdlib>

#define DTYPE unsigned long long
#define ull unsigned long long

/*** CUDA API error checking  ***/
#define cudaErrChk(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"CUDA assert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}



/****************************************************************
  *** Blocked reduction
  ****************************************************************/

/*** Kernel program ***/
__global__ void reduction_blocked (DTYPE* d_data, DTYPE* d_out, ull remain) {
    ull idx = blockIdx.x * blockDim.x + threadIdx.x;
    ull 
    DTYPE sum = 0;

    for (int s=blockDim.x>>1; s>0; s>>=1) {
        if (idx + s < remain) {
            d_data[out]+=d_data[out_s];
        }
        __syncthreads();
    }
    
    d_out[blockIdx.x] = sum;
}


/*** Host program ***/
ull run_kernel_basic (DTYPE* d_data, const ull num_data) {

    DTYPE* d_out;
    cudaErrChk (hipMalloc ((void**)&d_out, sizeof(DTYPE)*num_data));
    ull remain=num_data, next=0;

    dim3 threads (256);
    while (remain > 1) {
        next = remain/thread.x + remain%thread.x;

        dim3 blocks ((remain+threads.x-1)/threads.x);
        reduction_blocked<<<blocks, threads>>> (d_data, d_out, remain, next);
        cudaErrChk (hipMemcpy (d_data, d_out, next*sizeof(DYPT), hipMemcpyDeviceToDevice));
        cudaErrChk (hipDeviceSynchronize ())
        cudaErrChk (hipGetLastError() );
        remain = next;
    }


    cudaErrChk (hipFree (d_out));
    return remain;
}




/****************************************************************
  *** Basic reduction
  ****************************************************************/

/*** Kernel program ***/
__global__ void reduction (DTYPE* d_data, ull remain, ull next) {
    ull idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx + next < remain) {
        d_data[idx] += d_data[idx+next];
    }
}


/*** Host program ***/
ull run_kernel_basic (DTYPE* d_data, const ull num_data) {

    ull remain=num_data, next=0;
    while (remain > 1) {
        if (remain%2==0)
            next = remain/2;
        else
            next = remain/2 +1;

        dim3 threads (256);
        dim3 blocks ((next+threads.x-1)/threads.x);
        reduction<<<blocks, threads>>> (d_data, remain, next);
        cudaErrChk (hipDeviceSynchronize ())
        cudaErrChk (hipGetLastError() );
        remain = next;
    }

    return remain;
}





/****************************************************************
  *** Host program
  ****************************************************************/

DTYPE initial_data (DTYPE* data, const ull num_data) {
    DTYPE sum = 0;
    for (ull i=0; i<num_data; i++) {
        data[i] = rand()%5-2;
        sum += data[i];
    }
    return sum;
}


int main (int argc, char** argv) {

    /*** Program Configuration ***/
    const ull num_data = 5*1e+8;
    const size_t size_data = sizeof (ull) * num_data;
    printf("\n\n=======================================================================\n");
    printf("== Parallel integer reduction\n");
    printf("=======================================================================\n");
    printf("Number of int : %llu\n", num_data);
    printf("    size of mem : %.2f GB\n", size_data*1e-9);


    /*** Initialize variables ***/
    DTYPE* data = (DTYPE*) malloc (size_data);
    const DTYPE sum = initial_data (data, num_data);



    /*** Set CUDA Memory ***/
    DTYPE* d_data;
    cudaErrChk (hipMalloc ((void**)&d_data, size_data));
    cudaErrChk (hipMemcpy (d_data, data, size_data, hipMemcpyHostToDevice));
    cudaErrChk (hipDeviceSynchronize ())

    /*** Run kernel ***/
    run_kernel_basic (d_data, num_data);

    /*** Check result ***/
    DTYPE result;
    cudaErrChk (hipMemcpy (&result, d_data, sizeof (DTYPE), hipMemcpyDeviceToHost));
    
    printf("Check result ...\n");
    if (sum != result[0]) {
        printf("Err GT(%llu) != Pred(%llu)\n", sum, result);
    } else {
        printf("Pass GT(%llu) == Pred(%llu)\n", sum, result);
    }
    printf("=======================================================================\n\n");

    /*** Finalize program ***/
    cudaErrChk (hipFree (d_data));    
    free (data);

    return 0;
}
