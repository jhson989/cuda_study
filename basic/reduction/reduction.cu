

#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>

#define DTYPE unsigned long long
#define ull unsigned long long

/*** CUDA API error checking  ***/
#define cudaErrChk(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"CUDA assert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


/****************************************************************
  *** Kernel mode : 4
  *** Blocked shared half unrolled reduction
  ****************************************************************/

/*** Kernel program ***/
template <size_t blockSize>
__global__ void reduction_blocked_shared_half_unrolled (DTYPE* d_data, DTYPE* d_out, ull remain) {
    ull tidx = threadIdx.x;
    ull idx = blockIdx.x * (2*blockSize) + threadIdx.x;
    extern __shared__ DTYPE smem[];

    if (idx < remain) {
        if (idx+blockSize<remain)
            smem[tidx] = d_data[idx]+d_data[idx+blockSize];
        else
            smem[tidx] = d_data[idx];
    }
    __syncthreads();

    #pragma unroll
    for (ull s=blockSize>>1; s>0; s>>=1) {
        if (tidx<s && idx+s<remain) {
            smem[tidx]+=smem[tidx+s];
        }
        __syncthreads();
    }


    if (tidx == 0) {
        d_out[blockIdx.x] = smem[tidx];
    }
}



/*** Host program ***/
void run_kernel_blocked_shared_half_unrolled (DTYPE* d_data, const ull num_data) {

    DTYPE* d_out;
    cudaErrChk (hipMalloc ((void**)&d_out, sizeof(DTYPE)*num_data));
    ull remain=num_data, next=0;

    dim3 threads (128);
    const size_t size_smem = sizeof (DTYPE) * threads.x;
    while (remain > 1) {
        if (remain%threads.x==0)
            next = remain/(2*threads.x);
        else
            next = remain/(2*threads.x)+1;


        dim3 blocks ((remain+(2*threads.x)-1)/(2*threads.x));
        reduction_blocked_shared_half_unrolled<128><<<blocks, threads, size_smem>>> (d_data, d_out, remain);
        cudaErrChk (hipMemcpy (d_data, d_out, next*sizeof(DTYPE), hipMemcpyDeviceToDevice));
        cudaErrChk (hipDeviceSynchronize ())
        cudaErrChk (hipGetLastError() );
        
        remain = next;
    } 

    cudaErrChk (hipFree (d_out));
 
}






/****************************************************************
  *** Kernel mode : 3
  *** Blocked shared half reduction
  ****************************************************************/

/*** Kernel program ***/
__global__ void reduction_blocked_shared_half (DTYPE* d_data, DTYPE* d_out, ull remain) {
    ull tidx = threadIdx.x;
    ull idx = blockIdx.x * (2*blockDim.x) + threadIdx.x;
    extern __shared__ DTYPE smem[];

    if (idx < remain) {
        if (idx+blockDim.x<remain)
            smem[tidx] = d_data[idx]+d_data[idx+blockDim.x];
        else
            smem[tidx] = d_data[idx];
    }
    __syncthreads();

    for (ull s=blockDim.x>>1; s>0; s>>=1) {
        if (tidx<s && idx+s<remain) {
            smem[tidx]+=smem[tidx+s];
        }
        __syncthreads();
    }
    
    if (tidx == 0) {
        d_out[blockIdx.x] = smem[tidx];
    }
}



/*** Host program ***/
void run_kernel_blocked_shared_half (DTYPE* d_data, const ull num_data) {

    DTYPE* d_out;
    cudaErrChk (hipMalloc ((void**)&d_out, sizeof(DTYPE)*num_data));
    ull remain=num_data, next=0;

    dim3 threads (128);
    const size_t size_smem = sizeof (DTYPE) * threads.x;
    while (remain > 1) {
        if (remain%threads.x==0)
            next = remain/(2*threads.x);
        else
            next = remain/(2*threads.x)+1;


        dim3 blocks ((remain+(2*threads.x)-1)/(2*threads.x));
        reduction_blocked_shared_half<<<blocks, threads, size_smem>>> (d_data, d_out, remain);
        cudaErrChk (hipMemcpy (d_data, d_out, next*sizeof(DTYPE), hipMemcpyDeviceToDevice));
        cudaErrChk (hipDeviceSynchronize ())
        cudaErrChk (hipGetLastError() );
        
        remain = next;
    } 

    cudaErrChk (hipFree (d_out));
 
}






/****************************************************************
  *** Kernel mode : 2
  *** Blocked shared reduction
  ****************************************************************/

/*** Kernel program ***/
__global__ void reduction_blocked_shared (DTYPE* d_data, DTYPE* d_out, ull remain) {
    ull tidx = threadIdx.x;
    ull idx = blockIdx.x * blockDim.x + threadIdx.x;
    extern __shared__ DTYPE smem[];

    if (idx < remain) {
        smem[tidx] = d_data[idx];
    }
    __syncthreads();

    for (ull s=blockDim.x>>1; s>0; s>>=1) {
        if (tidx<s && idx+s<remain) {
            smem[tidx]+=smem[tidx+s];
        }
        __syncthreads();
    }
    
    if (tidx == 0) {
        d_out[blockIdx.x] = smem[tidx];
    }
}



/*** Host program ***/
void run_kernel_blocked_shared (DTYPE* d_data, const ull num_data) {

    DTYPE* d_out;
    cudaErrChk (hipMalloc ((void**)&d_out, sizeof(DTYPE)*num_data));
    ull remain=num_data, next=0;

    dim3 threads (128);
    const size_t size_smem = sizeof (DTYPE) * threads.x;
    while (remain > 1) {
        if (remain%threads.x==0)
            next = remain/threads.x;
        else
            next = remain/threads.x+1;


        dim3 blocks ((remain+threads.x-1)/threads.x);
        reduction_blocked_shared<<<blocks, threads, size_smem>>> (d_data, d_out, remain);
        cudaErrChk (hipMemcpy (d_data, d_out, next*sizeof(DTYPE), hipMemcpyDeviceToDevice));
        cudaErrChk (hipDeviceSynchronize ())
        cudaErrChk (hipGetLastError() );
        
        remain = next;
    } 

    cudaErrChk (hipFree (d_out));
 
}




/****************************************************************
  *** Kernel mode : 1
  *** Blocked reduction
  ****************************************************************/

/*** Kernel program ***/
__global__ void reduction_blocked (DTYPE* d_data, DTYPE* d_out, ull remain) {
    ull tidx = threadIdx.x;
    ull idx = blockIdx.x * blockDim.x + threadIdx.x;

    for (ull s=blockDim.x>>1; s>0; s>>=1) {
        if (tidx<s && idx+s < remain) {
            d_data[idx]+=d_data[idx+s];
        }
        __syncthreads();
    }
    
    if (tidx == 0) {
        d_out[blockIdx.x] = d_data[idx];
    }
}


/*** Host program ***/
void run_kernel_blocked (DTYPE* d_data, const ull num_data) {

    DTYPE* d_out;
    cudaErrChk (hipMalloc ((void**)&d_out, sizeof(DTYPE)*num_data));
    ull remain=num_data, next=0;

    dim3 threads (128);
    while (remain > 1) {
        if (remain%threads.x==0)
            next = remain/threads.x;
        else
            next = remain/threads.x+1;


        dim3 blocks ((remain+threads.x-1)/threads.x);
        reduction_blocked<<<blocks, threads>>> (d_data, d_out, remain);
        cudaErrChk (hipMemcpy (d_data, d_out, next*sizeof(DTYPE), hipMemcpyDeviceToDevice));
        cudaErrChk (hipDeviceSynchronize ())
        cudaErrChk (hipGetLastError() );
        
        remain = next;
    } 

    cudaErrChk (hipFree (d_out));
 
}




/****************************************************************
  *** Kernel mode : 0
  *** Basic reduction
  ****************************************************************/

/*** Kernel program ***/
__global__ void reduction (DTYPE* d_data, ull remain, ull next) {
    ull idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx + next < remain) {
        d_data[idx] += d_data[idx+next];
    }
}


/*** Host program ***/
void run_kernel_basic (DTYPE* d_data, const ull num_data) {

    ull remain=num_data, next=0;
    dim3 threads (128);

    while (remain > 1) {
        if (remain%2==0)
            next = remain/2;
        else
            next = remain/2 +1;

        dim3 blocks ((next+threads.x-1)/threads.x);
        reduction<<<blocks, threads>>> (d_data, remain, next);
        cudaErrChk (hipDeviceSynchronize ())
        cudaErrChk (hipGetLastError() );
        remain = next;
    }
}





/****************************************************************
  *** Host program
  ****************************************************************/

DTYPE initial_data (DTYPE* data, const ull num_data) {
    DTYPE sum = 0;
    for (ull i=0; i<num_data; i++) {
        data[i] = rand()%5;
        sum += data[i];
    }
    return sum;
}

int select_mode(const int argc, const char** argv) {

    int mode = 0;
    if (argc > 1)
        mode = atoi(argv[1]);

    switch (mode) {
        case 0:
            printf("Kernel mode : 0.Basic reduction\n");
            break;
        case 1:
            printf("Kernel mode : 1.Blocked reduction\n");
            break;
        case 2:
            printf("Kernel mode : 2.Blocked shared reduction\n");
            break;
        case 3:
            printf("Kernel mode : 3.Blocked shared half reduction\n");
            break;
        case 4:
            printf("Kernel mode : 4.Blocked shared half unrolled reduction\n");
            break;


        default:
            printf("Selected not implemented mode...\n");
            exit(1);
            break;
    }
    return mode;
}


int main (const int argc, const char** argv) {

    /*** Program Configuration ***/
    const ull num_data = 4*1e+8;
    const int loop_exe = 4;
    const size_t size_data = sizeof (ull) * num_data;

    printf("\n\n=======================================================================\n");
    printf("== Parallel DTYPE reduction\n");
    printf("=======================================================================\n");
    const int mode_kernel = select_mode(argc, argv);
    printf("Number of DTYPE : %llu\n", num_data);
    printf("    size of mem : %.2f GB\n", size_data*1e-9);


    /*** Initialize variables ***/
    DTYPE* data = (DTYPE*) malloc (size_data);
    const DTYPE sum = initial_data (data, num_data);
    float gops = 1e-9*num_data*loop_exe;
    hipEvent_t start, stop;
    float msec_total=0.0f, msec=0.0f;
    cudaErrChk (hipEventCreate(&start));
    cudaErrChk (hipEventCreate(&stop));


    /*** Set CUDA Memory ***/
    DTYPE* d_data;
    cudaErrChk (hipMalloc ((void**)&d_data, size_data));
    cudaErrChk (hipMemcpy (d_data, data, size_data, hipMemcpyHostToDevice));
    cudaErrChk (hipDeviceSynchronize ());


    /*** Run kernel ***/
    for (int loop=0; loop<loop_exe; loop++) {
        cudaErrChk (hipMemcpy (d_data, data, size_data, hipMemcpyHostToDevice));
        cudaErrChk (hipEventRecord(start, NULL));
        switch (mode_kernel) {
            case 0:
                run_kernel_basic (d_data, num_data);
                break;
            case 1:
                run_kernel_blocked (d_data, num_data);
                break;
            case 2:
                run_kernel_blocked_shared (d_data, num_data);
                break;
            case 3:
                run_kernel_blocked_shared_half (d_data, num_data);
                break;
            case 4:
                run_kernel_blocked_shared_half_unrolled (d_data, num_data);
                break;
            default:
                printf("Not implemented\n");
                exit(1);
                break;
        }
        cudaErrChk (hipEventRecord(stop, NULL));
        cudaErrChk (hipEventSynchronize(stop));
        cudaErrChk (hipEventElapsedTime(&msec, start, stop));
        msec_total += msec;
    }

    /*** Check result ***/
    DTYPE result;
    cudaErrChk (hipMemcpy (&result, d_data, sizeof (DTYPE), hipMemcpyDeviceToHost));

    printf("    Total number of add inst. : %.2f Gops\n", gops);
    printf("    Elaped time: %.4f msec\n", msec_total);
    printf("    GFLOPS : %.4f gflops [Avg. of %d time(s)]\n", gops/(msec_total*1e-3), loop_exe); 


    printf("Check result ...\n");
    if (sum != result) {
        printf("    [Err] GT(%llu) != Pred(%llu)\n", sum, result);
    } else {
        printf("    [Pass] GT(%llu) == Pred(%llu)\n", sum, result);
    }
    printf("=======================================================================\n\n");

    /*** Finalize program ***/
    cudaErrChk (hipFree (d_data));    
    free (data);

    return 0;
}
