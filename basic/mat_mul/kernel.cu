#include "hip/hip_runtime.h"

#include <cstdio>
#include "kernel.hpp"


__global__ void matmul_basic(const float* A, const float* B, float* C, const int len) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < len*len) {
        int i=idx/len, j=idx%len;
        float sum=0;

        for (int k=0; k<len; k++) {
            sum += A[i*len+k]*B[k*len+j];
        }
        C[idx]=sum;
    }
}
