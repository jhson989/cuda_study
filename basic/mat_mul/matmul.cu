
#include <cstdio>
#include <cstdlib>
#include <random>
#include <sys/time.h>
#include "kernel.cuh"

#define cudaErrChk(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"CUDA assert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int loop_exe = 1;

void matmul_serial (const float *A, const float *B, float *C, const int len) {
    printf("[CPU] Kernel start..\n");
    

    timeval st, ed;
    gettimeofday(&st, NULL);
    // Main body
    for (int i=0; i<len; i++) {
        for (int j=0; j<len; j++) {
            float sum = 0;
            for (int k=0; k<len; k++) {
                sum += A[i*len+k]*B[k*len+j];
            }
            C[i*len+j] = sum;
        }
    }
    // End of main body
    gettimeofday(&ed, NULL);

    float time = (ed.tv_sec - st.tv_sec) + ((ed.tv_usec-st.tv_usec)*1e-6);
    float gops = 1.0*len*len*len*1e-9;
    printf("    Total number of floating point multiplications : %.2fGops\n", gops);
    printf("    Elaped time: %.4f\n", time);
    printf("    GFLOPS : %.4f [Avg. of %d time(s)]\n", gops*loop_exe/time, loop_exe); 

}


void matmul_cuda_basic (const float *A, const float *B, float *C, const int len) {

    /***
      CUDA implementataion without any optimization methods
      **/
    const int num_threads = 16;
    const dim3 dim_threads(num_threads, num_threads);
    const dim3 dim_blocks((len+num_threads-1)/num_threads, (len+num_threads-1)/num_threads);
    printf("[GPU] Basic kernel start..\n");
    printf("    Grid size: [(%d, %d), (%d, %d)]\n", dim_blocks.y, dim_blocks.x, dim_threads.y, dim_threads.x);


    /*** Memcpy H to D ***/
    float *d_A, *d_B, *d_C;
    cudaErrChk (hipMalloc ((void **)&d_A, sizeof(float)*len*len));
    cudaErrChk (hipMalloc ((void **)&d_B, sizeof(float)*len*len));
    cudaErrChk (hipMalloc ((void **)&d_C, sizeof(float)*len*len));
    cudaErrChk (hipMemcpy (d_A, A, sizeof(float)*len*len, hipMemcpyHostToDevice));
    cudaErrChk (hipMemcpy (d_B, B, sizeof(float)*len*len, hipMemcpyHostToDevice));
    

    float gops = 1.0*len*len*len*1e-9*loop_exe;
    float msec_total = 0.0f;
    hipEvent_t start, stop;
    cudaErrChk(hipEventCreate(&start));
    cudaErrChk(hipEventCreate(&stop));
    cudaErrChk(hipEventRecord(start, NULL));
    // Main body
    for (int i=0; i<loop_exe; i++) {
        matmul_basic<<<dim_blocks, dim_threads>>>(d_A, d_B, d_C, len);
        cudaErrChk (hipDeviceSynchronize ())
        cudaErrChk( hipGetLastError() );
    }
    // End of main body
    cudaErrChk(hipEventRecord(stop, NULL));
    cudaErrChk(hipEventSynchronize(stop));
    cudaErrChk(hipEventElapsedTime(&msec_total, start, stop));


    printf("    Total number of floating point multiplications : %.2f Gops\n", gops);
    printf("    Elaped time: %.4f msec\n", msec_total);
    printf("    GFLOPS : %.4f [Avg. of %d time(s)]\n", gops/(msec_total*1e-3), loop_exe); 

    cudaErrChk (hipMemcpy(C, d_C, sizeof(float)*len*len, hipMemcpyDeviceToHost));
    cudaErrChk (hipDeviceSynchronize ())
    cudaErrChk (hipFree (d_A));
    cudaErrChk (hipFree (d_B));
    cudaErrChk (hipFree (d_C));

}



void matmul_cuda_shared (const float *A, const float *B, float *C, const int len) {

    /***
      CUDA implementataion without any optimization methods
      **/
    const int len_tile = 16;
    const dim3 dim_threads(len_tile, len_tile);
    const dim3 dim_blocks((len+len_tile-1)/len_tile, (len+len_tile-1)/len_tile);
    const int size_smem = 2*sizeof(float)*len_tile*len_tile;
    printf("[GPU] Kernel with shared memory start..\n");
    printf("    Grid size: [(%d, %d), (%d, %d)]\n", dim_blocks.y, dim_blocks.x, dim_threads.y, dim_threads.x);
    printf("    Shared mem size: %.2fKB\n", size_smem/1024.0);


    /*** Memcpy H to D ***/
    float *d_A, *d_B, *d_C;
    cudaErrChk (hipMalloc ((void **)&d_A, sizeof(float)*len*len));
    cudaErrChk (hipMalloc ((void **)&d_B, sizeof(float)*len*len));
    cudaErrChk (hipMalloc ((void **)&d_C, sizeof(float)*len*len));
    cudaErrChk (hipMemcpy (d_A, A, sizeof(float)*len*len, hipMemcpyHostToDevice));
    cudaErrChk (hipMemcpy (d_B, B, sizeof(float)*len*len, hipMemcpyHostToDevice));
    

    float gops = 1.0*len*len*len*1e-9*loop_exe;
    float msec_total = 0.0f;
    hipEvent_t start, stop;
    cudaErrChk(hipEventCreate(&start));
    cudaErrChk(hipEventCreate(&stop));
    cudaErrChk(hipEventRecord(start, NULL));
    // Main body
    for (int i=0; i<loop_exe; i++) {
        matmul_tiled<<<dim_blocks, dim_threads, size_smem>>>(d_A, d_B, d_C, len, len_tile);
        cudaErrChk (hipDeviceSynchronize ())
        cudaErrChk( hipGetLastError() );
    }
    // End of main body
    cudaErrChk(hipEventRecord(stop, NULL));
    cudaErrChk(hipEventSynchronize(stop));
    cudaErrChk(hipEventElapsedTime(&msec_total, start, stop));

    printf("    Total number of floating point multiplications : %.2f Gops\n", gops);
    printf("    Elaped time: %.4f msec\n", msec_total);
    printf("    GFLOPS : %.4f [Avg. of %d time(s)]\n", gops/(msec_total*1e-3), loop_exe); 

    cudaErrChk (hipMemcpy(C, d_C, sizeof(float)*len*len, hipMemcpyDeviceToHost));
    cudaErrChk (hipDeviceSynchronize ())
    cudaErrChk (hipFree (d_A));
    cudaErrChk (hipFree (d_B));
    cudaErrChk (hipFree (d_C));

}


void matmul_cuda_shared_transposed (const float *A, const float *B, float *C, const int len) {

    /***
      CUDA implementataion without any optimization methods
      **/
    const int len_tile = 16;
    const dim3 dim_threads(len_tile, len_tile);
    const dim3 dim_blocks((len+len_tile-1)/len_tile, (len+len_tile-1)/len_tile);
    const int size_smem = 2*sizeof(float)*len_tile*len_tile;
    printf("[GPU] Kernel with shared transposed start..\n");
    printf("    Grid size: [(%d, %d), (%d, %d)]\n", dim_blocks.y, dim_blocks.x, dim_threads.y, dim_threads.x);
    printf("    Shared mem size: %.2fKB\n", size_smem/1024.0);


    /*** Memcpy H to D ***/
    float *d_A, *d_A_T, *d_B, *d_C;
    cudaErrChk (hipMalloc ((void **)&d_A, sizeof(float)*len*len));
    cudaErrChk (hipMalloc ((void **)&d_A_T, sizeof(float)*len*len));
    cudaErrChk (hipMalloc ((void **)&d_B, sizeof(float)*len*len));
    cudaErrChk (hipMalloc ((void **)&d_C, sizeof(float)*len*len));
    cudaErrChk (hipMemcpy (d_A, A, sizeof(float)*len*len, hipMemcpyHostToDevice));
    transpose<<<dim_blocks, dim_threads>>>(d_A, d_A_T, len);
    cudaErrChk (hipMemcpy (d_B, B, sizeof(float)*len*len, hipMemcpyHostToDevice));
    

    float gops = 1.0*len*len*len*1e-9*loop_exe;
    float msec_total = 0.0f;
    hipEvent_t start, stop;
    cudaErrChk(hipEventCreate(&start));
    cudaErrChk(hipEventCreate(&stop));
    cudaErrChk(hipEventRecord(start, NULL));
    // Main body
    for (int i=0; i<loop_exe; i++) {
        matmul_tiled_transposed<<<dim_blocks, dim_threads, size_smem>>>(d_A_T, d_B, d_C, len, len_tile);
        cudaErrChk (hipDeviceSynchronize ())
        cudaErrChk( hipGetLastError() );
    }
    // End of main body
    cudaErrChk(hipEventRecord(stop, NULL));
    cudaErrChk(hipEventSynchronize(stop));
    cudaErrChk(hipEventElapsedTime(&msec_total, start, stop));

    printf("    Total number of floating point multiplications : %.2f Gops\n", gops);
    printf("    Elaped time: %.4f msec\n", msec_total);
    printf("    GFLOPS : %.4f [Avg. of %d time(s)]\n", gops/(msec_total*1e-3), loop_exe); 

    cudaErrChk (hipMemcpy(C, d_C, sizeof(float)*len*len, hipMemcpyDeviceToHost));
    cudaErrChk (hipDeviceSynchronize ())
    cudaErrChk (hipFree (d_A));
    cudaErrChk (hipFree (d_A_T));
    cudaErrChk (hipFree (d_B));
    cudaErrChk (hipFree (d_C));

}

/****************************************
  ************** Host Code **************
  ****************************************/

void h_initialize(float *mem, const int len) {
    for (int i=0; i<len; i++) {
        for (int j=0; j<len; j++) {
            mem[i*len+j] = (float)(rand()%1000);
        }
    }
}

bool h_test(const float *A, const float *B, const float *C, const int len) {
    printf("[TEST] Test start..\n");
    for (int i=0; i<len; i++) {
        for (int j=0; j<len; j++) {
            float sum = 0;
            for (int k=0; k<len; k++) {
                sum += A[i*len+k]*B[k*len+j];
            }
            if (sum != C[i*len+j]){
                printf("C[%d][%d] = %.f != %f\n", i, j, C[i*len+j], sum);
                return false;
            }
        }
    }
    return true;
}


int main(int argc, char** argv) {

    /*** Program configuration ***/
    printf("\n============================================\n");
    printf("Matrix multiplication\n");
    printf("    A * B = C\n");
    printf("    arg : ./matmul [len] [Test:0,1] [loop_exe]\n");
    printf("============================================\n\n");
    int len = (int)1e+3;
    if (argc >= 2) 
        len = atoi(argv[1]);
    if (argc >= 4) 
        loop_exe = atoi(argv[3]);
    srand(0);

    /*** Data initialize ***/
    float *A = (float *) malloc (len*len*sizeof(float));
    float *B = (float *) malloc (len*len*sizeof(float));
    float *C = (float *) calloc (len*len,sizeof(float));
    h_initialize(A, len);
    h_initialize(B, len);
    printf("[Mem] Size of a matrix : [%d, %d]\n", len, len);
    printf("[Mem] Total size of matrices : %.3fGB\n", 3.0*len*len*sizeof(float)*1e-9);


    /*** Run a matmul ***/
//    matmul_serial (A, B, C, len);
    matmul_cuda_basic (A, B, C, len);
    matmul_cuda_shared (A, B, C, len);
    matmul_cuda_shared_transposed (A, B, C, len);

    /*** Test the result ***/
    if (argc >= 3 && atoi(argv[2]) == 1) {
        if (h_test (A, B, C, len) == true) {
            printf("    Test passed\n");
        } else {
            printf("    [ERR] Test failed!!\n");
        }
    } else {
        printf("[TEST] Test skipped..\n");
    }

    /*** Finalize ***/
    free (A);
    free (B);
    free (C);


    printf("============================================\n\n");
    return 0;
}


